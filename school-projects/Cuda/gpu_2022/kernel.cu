#include "hip/hip_runtime.h"
#define BLOCK_SIZE 32

__global__ void calculate(sGalaxy A, sGalaxy B, int n, float* out){
    
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float Asx[BLOCK_SIZE];
    __shared__ float Asy[BLOCK_SIZE];
    __shared__ float Asz[BLOCK_SIZE];
    __shared__ float Bsx[BLOCK_SIZE];
    __shared__ float Bsy[BLOCK_SIZE];
    __shared__ float Bsz[BLOCK_SIZE];

    if(i > n-1) return;
    
    float tmp = 0.0f;

    Asx[threadIdx.x] = A.x[threadIdx.x];
    Asy[threadIdx.x] = A.y[threadIdx.x];
    Asz[threadIdx.x] = A.z[threadIdx.x];
    Bsx[threadIdx.x] = B.x[threadIdx.x];
    Bsy[threadIdx.x] = B.y[threadIdx.x];
    Bsz[threadIdx.x] = B.z[threadIdx.x];
    __syncthreads();

    for(int idx = threadIdx.x + 1; idx < BLOCK_SIZE && idx < n; idx++)
    {   
        float da = sqrt((Asx[idx]-A.x[i])*(Asx[idx]-A.x[i])
            + (Asy[idx]-A.y[i])*(Asy[idx]-A.y[i])
            + (Asz[idx]-A.z[i])*(Asz[idx]-A.z[i]));
        float db = sqrt((Bsx[idx]-B.x[i])*(Bsx[idx]-B.x[i])
            + (Bsy[idx]-B.y[i])*(Bsy[idx]-B.y[i])
            + (Bsz[idx]-B.z[i])*(Bsz[idx]-B.z[i]));
        tmp += (da-db) * (da-db);
    }

    for(int b = blockIdx.x + 1; b < (n / BLOCK_SIZE)-1; b++)
    {
        Asx[threadIdx.x] = A.x[threadIdx.x + b * BLOCK_SIZE];
        Asy[threadIdx.x] = A.y[threadIdx.x + b * BLOCK_SIZE];
        Asz[threadIdx.x] = A.z[threadIdx.x + b * BLOCK_SIZE];
        Bsx[threadIdx.x] = B.x[threadIdx.x + b * BLOCK_SIZE];
        Bsy[threadIdx.x] = B.y[threadIdx.x + b * BLOCK_SIZE];
        Bsz[threadIdx.x] = B.z[threadIdx.x + b * BLOCK_SIZE];
        __syncthreads();
        for(int idx = 0 ; idx < BLOCK_SIZE; idx++)
        {
            float da = sqrt((Asx[idx]-A.x[i])*(Asx[idx]-A.x[i])
                + (Asy[idx]-A.y[i])*(Asy[idx]-A.y[i])
                + (Asz[idx]-A.z[i])*(Asz[idx]-A.z[i]));
            float db = sqrt((Bsx[idx]-B.x[i])*(Bsx[idx]-B.x[i])
                + (Bsy[idx]-B.y[i])*(Bsy[idx]-B.y[i])
                + (Bsz[idx]-B.z[i])*(Bsz[idx]-B.z[i]));
            tmp += (da-db) * (da-db);
        }	
	}

    Asx[threadIdx.x] = A.x[threadIdx.x + ((n / BLOCK_SIZE) * BLOCK_SIZE)];
    Asy[threadIdx.x] = A.y[threadIdx.x + ((n / BLOCK_SIZE) * BLOCK_SIZE)];
    Asz[threadIdx.x] = A.z[threadIdx.x + ((n / BLOCK_SIZE) * BLOCK_SIZE)];
    Bsx[threadIdx.x] = B.x[threadIdx.x + ((n / BLOCK_SIZE) * BLOCK_SIZE)];
    Bsy[threadIdx.x] = B.y[threadIdx.x + ((n / BLOCK_SIZE) * BLOCK_SIZE)];
    Bsz[threadIdx.x] = B.z[threadIdx.x + ((n / BLOCK_SIZE) * BLOCK_SIZE)];
    __syncthreads();

    for(int idx = 0; idx < BLOCK_SIZE && 
    (idx + (((n / BLOCK_SIZE)) * BLOCK_SIZE) ) < n; idx++)
    {
        float da = sqrt((Asx[idx]-A.x[i])*(Asx[idx]-A.x[i])
            + (Asy[idx]-A.y[i])*(Asy[idx]-A.y[i])
            + (Asz[idx]-A.z[i])*(Asz[idx]-A.z[i]));
        float db = sqrt((Bsx[idx]-B.x[i])*(Bsx[idx]-B.x[i])
            + (Bsy[idx]-B.y[i])*(Bsy[idx]-B.y[i])
            + (Bsz[idx]-B.z[i])*(Bsz[idx]-B.z[i]));
        tmp += (da-db) * (da-db);
    }
    out[i] = tmp;
}



float solveGPU(sGalaxy A, sGalaxy B, int n) {
    int vector_size = n;
    float *d_diff, *diff;

    hipHostMalloc((void **) &diff, vector_size * sizeof(float)); 
    
    if (hipMalloc((void **) &d_diff, vector_size * sizeof(d_diff[0])) != hipSuccess) {
		fprintf(stderr, "Device memory allocation error!\n");
		hipFree(d_diff);
	}

    int blocksPerGrid = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    calculate<<< blocksPerGrid,  BLOCK_SIZE>>>(A, B, n, d_diff);
    

    hipMemcpy(diff, d_diff, vector_size * sizeof(d_diff[0]), hipMemcpyDeviceToHost);
    
    float diff_result = 0.0f;
    
    for (int i = 0; i < vector_size; i++){
        diff_result += diff[i];
    }  

    if (d_diff) hipFree(d_diff);
	if (diff) hipHostFree(diff);

    return sqrt(1/((float)n*((float)n-1)) * diff_result);
}